
#include <hip/hip_runtime.h>
#include <stdio.h>

#define cudaCheckErrors(msg) \
do { \
  hipError_t __err = hipGetLastError(); \
  if (__err != hipSuccess) { \
    fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
    msg, hipGetErrorString(__err), \
    __FILE__, __LINE__); \
    fprintf(stderr, "*** FAILED - ABORTING\n"); \
    exit(1); \
  } \
} while (0)

__global__ void testkernel(int *data, int size){

  for (int i = 1; i < size; i++) data[0] += data[i];
}
extern "C" {
  int cudatestfunc(int *data, int size){

    int *d_data;
    hipMalloc(&d_data, size*sizeof(int));
    hipMemcpy(d_data, data, size*sizeof(int), hipMemcpyHostToDevice);
    testkernel<<<1,1>>>(d_data, size);
    int result;
    hipMemcpy(&result, d_data, sizeof(int), hipMemcpyDeviceToHost);
    cudaCheckErrors("cuda error");
    return result;
  }
}
